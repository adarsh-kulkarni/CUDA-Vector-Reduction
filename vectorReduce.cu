#include "hip/hip_runtime.h"


// Vector Reduction
//

// Includes
#include <stdio.h>
#include <cutil_inline.h>



// Input Array Variables
float* h_In = NULL;
float* d_In = NULL;

// Output Array
float* h_Out = NULL;
float* d_Out = NULL;

// Variables to change
int GlobalSize = 50000;
int BlockSize = 32;

//Timer Variables
unsigned int timer_mem=0;
unsigned int timer_total=0;
unsigned int timer_GPU=0;
unsigned int timer_CPU=0;

unsigned int total_mem=0;


// Functions
void Cleanup(void);
void RandomInit(float*, int);
void PrintArray(float*, int);
float CPUReduce(float*, int);
void ParseArguments(int, char**);




// Device code
__global__ void VecReduce(float* g_idata, float* g_odata, int N)
{
  // shared memory size declared at kernel launch
  extern __shared__ float sdata[]; 

  unsigned int tid = threadIdx.x; 
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x; 

  // For thread ids greater than data space
  if (globalid < N) {
     sdata[tid] = g_idata[globalid]; 
  }
  else {
     sdata[tid] = 0;  // Case of extra threads above N
  }

  // each thread loads one element from global to shared mem
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x / 2; s > 0; s = s >> 1) {
     if (tid < s) { 
         sdata[tid] = sdata[tid] + sdata[tid+ s];
     }
     __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0)  {
    g_odata[blockIdx.x] = sdata[0];
    //atomicAdd(&g_odata[blockIdx.x],sdata[0]);
  }

  
}


// Host code
int main(int argc, char** argv)
{
    ParseArguments(argc, argv);

    int N = GlobalSize;
    printf("Vector reduction: Input size %d\n", N);
    
    size_t in_size = N * sizeof(float);
    float CPU_result = 0.0, GPU_result = 0.0;

    // Allocate input vectors h_In and h_B in host memory
    h_In = (float*)malloc(in_size);
    if (h_In == 0) 
      Cleanup();

    // Initialize input vectors
    RandomInit(h_In, N);

    // Set the kernel arguments
    int threadsPerBlock = BlockSize;
    int sharedMemSize = threadsPerBlock * sizeof(float);
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; 
    
    size_t out_size = blocksPerGrid * sizeof(float);
  
    printf("Vector reduction: Block size %d\n", BlockSize); 
    
    // Allocate host output
    h_Out = (float*)malloc(out_size);
    if (h_Out == 0) 
      Cleanup();

    // STUDENT: CPU computation - time this routine for base comparison
    CPU_result = CPUReduce(h_In, N);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_In, in_size) );
    cutilSafeCall( hipMalloc((void**)&d_Out, out_size) );

    //Initialize timers to zero
    cutilCheckError(cutCreateTimer(&timer_mem));
    cutilCheckError(cutCreateTimer(&timer_total));
    cutilCheckError(cutCreateTimer(&timer_GPU));
    cutilCheckError(cutCreateTimer(&timer_CPU));
  


    //Start the timer for memory and total execution
    cutilCheckError(cutStartTimer(timer_mem));
    cutilCheckError(cutStartTimer(timer_total));

    // STUDENT: Copy h_In from host memory to device memory
    hipMemcpy(d_In, h_In, in_size, hipMemcpyHostToDevice);

    //Stop the Memory timer   
    cutilCheckError(cutStopTimer(timer_mem));
    
    //Start the GPU timer
    cutilCheckError(cutStartTimer(timer_GPU));
    
    // Invoke kernel
    VecReduce<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_In, d_Out, N);
    cutilCheckMsg("kernel launch failure");
    cutilSafeCall( hipDeviceSynchronize() ); // Have host wait for kernel
    cutilCheckError(cutStopTimer(timer_GPU));

    //Start the second memory timer
    cutilCheckError(cutStartTimer(timer_mem));
    
    // STUDENT: copy results back from GPU to the h_Out
    hipMemcpy(h_Out, d_Out, out_size, hipMemcpyDeviceToHost); 
    
    //Stop the second timer 
    cutilCheckError(cutStopTimer(timer_mem));
    cutilCheckError(cutStopTimer(timer_total));
    
 
    // STUDENT: Perform the CPU addition of partial results
    // update variable GPU_result

    //Start the timer for CPU adding partial sums
    cutilCheckError(cutStartTimer(timer_CPU));
   
    int j=0;

    for(j = 0; j < blocksPerGrid; j++){ 
     
    GPU_result = GPU_result + h_Out[j];

    }

    //Stop the CPU timer
    cutilCheckError(cutStopTimer(timer_CPU));
    

    //GPU_result = cpusum;
    //GPU_result = *h_Out;  
  

    // STUDENT Check results to make sure they are the same
    printf("CPU results : %f\n", CPU_result);
    printf("GPU results : %f\n", GPU_result);

    //Print the Timer values
    printf("GPU Execution Time: %f (ms) \n", cutGetTimerValue(timer_GPU));
    printf("Total Memory Transfer time : %f (ms) \n",cutGetTimerValue(timer_mem));
    printf("CPU Execution Time: %f (ms) \n", cutGetTimerValue(timer_CPU));
    printf("Overall Execution Time (Memory + GPU): %f (ms) \n", cutGetTimerValue(timer_total));
 
    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_In)
        hipFree(d_In);
    if (d_Out)
        hipFree(d_Out);

    // Free host memory
    if (h_In)
        free(h_In);
    if (h_Out)
        free(h_Out);

    cutilCheckError(cutDeleteTimer(timer_GPU));
    cutilCheckError(cutDeleteTimer(timer_mem));

    cutilCheckError(cutDeleteTimer(timer_total));
    cutilCheckError(cutDeleteTimer(timer_CPU));
    
        
    cutilSafeCall( hipDeviceReset() );
    
    exit(0);
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; i++)
        data[i] = rand() / (float)RAND_MAX;
}

void PrintArray(float* data, int n)
{
    for (int i = 0; i < n; i++)
        printf("[%d] => %f\n",i,data[i]);
}

float CPUReduce(float* data, int n)
{
  float sum = 0.0;
    for (int i = 0; i < n; i++)
        sum = sum + data[i];

  return sum;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--size") == 0 || strcmp(argv[i], "-size") == 0) {
                  GlobalSize = atoi(argv[i+1]);
		  i = i + 1;
        }
        if (strcmp(argv[i], "--blocksize") == 0 || strcmp(argv[i], "-blocksize") == 0) {
                  BlockSize = atoi(argv[i+1]); 
		  i = i + 1;
	}
    }
}
